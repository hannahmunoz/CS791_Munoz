#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <time.h>
#include <fstream> 
#include <string>

#include "kthnearestneighbor.h"

const int GLOBAL_CONST_ROW = 161;
const int GLOBAL_CONST_COL = 128;

void fileIn (std::string name, float* parsedCSV);

int main (int argc, char* argv[]){

	hipDeviceProp_t prop;
 	hipGetDeviceProperties( &prop, 0 );

	//variables
	int blockDim, threadDim;

	// get inputs
	if (argc < 4){
		std::cout << "Not enough arguments. <<filename>> << block dimension>> << thread dimension>>" << std::endl; 
		return 1;
	}
	else{
	       blockDim = atoi(argv [2]);
	       threadDim = atoi(argv [3]);
	}
	if (blockDim*threadDim <  sqrt(GLOBAL_CONST_ROW)){
		std::cout << "error: blocks and threads must cover the input file" << std::endl;
		std::cout << "must equal " << (int)sqrt(GLOBAL_CONST_ROW) << std::endl;
		return 1;
	} 


	srand(1);

	// initalize more varaibles
	dim3 grid (blockDim, blockDim);

	dim3 block (threadDim , threadDim );

	//create vector
	float* parsedCSV;
	float* results;
	float* kresults;

	//alloc memory
	hipMallocManaged( (void**)&parsedCSV, GLOBAL_CONST_ROW * GLOBAL_CONST_COL * sizeof(float) );
	hipMallocManaged( (void**)&results, GLOBAL_CONST_ROW * sizeof(float) );
	hipMallocManaged( (void**)&kresults, GLOBAL_CONST_ROW * sizeof(float) );


	fileIn (argv[1], parsedCSV);

	/*for (int i = 0; i < GLOBAL_CONST_ROW; i++){
		for (int j = 0; j < GLOBAL_CONST_COL; j++){
			printf ("%.02f ",parsedCSV[i*(GLOBAL_CONST_COL)+j] );
		}
		printf ("\n");
	}*/


	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );

	for (int i = 0; i < GLOBAL_CONST_ROW; i++){
		if (isnan(parsedCSV[i*(GLOBAL_CONST_COL)])){
			kDistance <<<grid, block>>> (parsedCSV, i, results, kresults);
		}
	
	}

	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );

	for (int i = 0; i < GLOBAL_CONST_ROW; i++){
		if (kresults[i] != 0.00){
			printf ("Row, %d,	k, %.02f\n", i, kresults[i]);
		}
	}

	//for testing output

 	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	//dealloc memory
    	hipEventDestroy( start );
        hipEventDestroy( end );
	hipFree (parsedCSV);
	hipFree (results);
	hipFree (kresults);

}

void fileIn (std::string name, float* parsedCSV){
	
	std::ifstream file (name.c_str());
	std::string s;

	// discard metadata on top
	if (file.good()){
		for (int i = 0; i < 9; i++){
			getline (file, s);
		}
	
		getline(file, s, ',');
		int iter = 1;
		while (getline(file, s, ',')) {
			if ( iter % 128 == 1 && rand() % 10 == 1){
				parsedCSV[iter-1] = NAN;

			}
			else{
				parsedCSV[iter-1] = atof (s.c_str());
			}
			iter++;
		}
	}
	file.close();

}
