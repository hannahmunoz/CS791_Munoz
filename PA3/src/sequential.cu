
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h> 
#include <cmath>
#include <stdio.h>
#include <string>
#include <fstream> 
#include <vector>
#include <time.h>


const int GLOBAL_CONST_ROW = 161;
const int GLOBAL_CONST_COL = 128;

int main (int argc, char* argv[]){
	//variables
	if (argc < 2){
		std::cout << "No file name given" << std::endl;
		return 1;
	}
	
	srand(time(NULL));

	std::vector <std::vector <float> > data;	

	// read in file
	std::ifstream file (argv[1]);
	std::string s;
	std::vector<float> parsedCSV;

	// discard metadata on top
	if (file.good()){
		for (int i = 0; i < 9; i++){
			getline (file, s);
		}
	
		getline(file, s, ',');
		int iter = 1;
		while (getline(file, s, ',')) {
			if ( iter % 129 == 1){
				if ( rand() % 10 == 1){
					parsedCSV.push_back(0.00);

				}
				else{
					parsedCSV.push_back(atof (s.c_str()));
				}
			}
			else{
				parsedCSV.push_back(atof (s.c_str()));
			}
		}
	}
	file.close();

	for (int i = 0; i < GLOBAL_CONST_ROW; i++){
		for (int j = 0; j < GLOBAL_CONST_COL; j++){
			std::cout << parsedCSV[i*(GLOBAL_CONST_COL)+j] << " ";
		}
		std::cout << std::endl << std::endl;
	}


	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );
	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );


	

	// read out file

	//output results
	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	return 0;

}
