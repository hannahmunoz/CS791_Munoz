#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <cmath>

#include "multiply.h"

bool isSquare(int num){	return (floor (sqrt(num)) == sqrt(num));}

int main (int argc, char* argv[]){
	
	//variables
	int matDim, blockDim, threadDim;

	// get inputs
	if (argc < 4){
		std::cout << "Not enough arguments. <<matrix dimension>> << block dimension>> << thread dimension>>" << std::endl; 
		return 1;
	}
	else{
	       matDim = atoi (argv [1]);
	       blockDim = atoi(argv [2]);
	       threadDim = atoi(argv [3]);
	}

	hipDeviceProp_t prop;
 	hipGetDeviceProperties( &prop, 0 );

	// bounds checking
	if ( matDim <=0 || matDim >= 32000){
		std::cout << "Matrix dimension not valid. Must be between 0 and 32000." << std::endl;
		return 1;
	}
	if ( blockDim <=0 || blockDim >= 25000 ){
		std::cout << "Block dimension not valid. Must be between 0 and 25000." << std::endl;
		return 1;
	}
	if ( threadDim <=0 || threadDim > sqrt(prop.maxThreadsPerBlock) ){
		std::cout << "Thread dimension not valid. Must be between 0 and " << sqrt(prop.maxThreadsPerBlock)  << "." << std::endl;
		return 1;
	}
	/*if ( blockDim * threadDim != matDim){
		std::cout << "Not enough/too many blocks and threads for given matrix dimensions" << std::endl;
		return 1;
	}*/

	// initalize more varaibles
	dim3 grid (blockDim, blockDim);
	dim3 block (threadDim, threadDim);

	//create arrays
	float *MatA, *MatB, *MatC;

	//alloc memory
	hipMallocManaged( (void**)&MatA, (float)pow(matDim, 2) * sizeof(float) );
	hipMallocManaged( (void**)&MatB, (float)pow(matDim, 2) * sizeof(float) );
	hipMallocManaged( (void**)&MatC, (float)pow(matDim, 2) * sizeof(float) );

	for (int i=0; i < (int)pow(matDim, 2); i++) {
 		MatA[i] = (float) i;
 		MatB[i] = (float) i;
 	}

	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );

	//send to GPU
	//hipMemcpy (a, MatA, (float)pow(matDim, 2) * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy (b, MatB, (float)pow(matDim, 2) * sizeof(float), hipMemcpyHostToDevice);

	//multiply
	multiply <<<grid, block>>> (MatA, MatB, MatC);

	// get result from GPU
	//hipMemcpy (MatC, c, (float)pow(matDim, 2) * sizeof(float), hipMemcpyDeviceToHost );

	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );

	//for testing output
	for (int i = 0; i < matDim; i++){
		for (int j = 0; j < matDim; j++){
			printf ("%.2f \t", MatC[(i*matDim)+j]);
			//std::cout << MatC[(i*matDim)+j] << "\t";
		}
		std::cout << std::endl;
	}

 	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	//dealloc memory
    	hipEventDestroy( start );
        hipEventDestroy( end );
	hipFree (MatA);
	hipFree (MatB);
	hipFree (MatC);
}
