#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <cmath>

#include "add.h"

bool isSquare(int num){	return (floor (sqrt(num)) == sqrt(num));}

int main (int argc, char* argv[]){
	
	//variables
	int matDim, blockDim, threadDim;

	// get inputs
	if (argc < 4){
		std::cout << "Not enough arguments. <<matrix dimension>> << block dimension>> << thread dimension>>" << std::endl; 
		return 1;
	}
	else{
	       matDim = atoi (argv [1]);
	       blockDim = atoi(argv [2]);
	       threadDim = atoi(argv [3]);
	}

	hipDeviceProp_t prop;
 	hipGetDeviceProperties( &prop, 0 );

	// bounds checking
	if ( matDim <=0 || matDim >= 32000){
		std::cout << "Matrix dimension not valid. Must be between 0 and 32000." << std::endl;
		return 1;
	}
	if ( blockDim <=0 || blockDim >= 25000 ){
		std::cout << "Block dimension not valid. Must be between 0 and 25000." << std::endl;
		return 1;
	}
	if ( threadDim <=0 || threadDim > sqrt(prop.maxThreadsPerBlock) ){
		std::cout << "Thread dimension not valid. Must be between 0 and " << sqrt(prop.maxThreadsPerBlock)  << "." << std::endl;
		return 1;
	}
	if ( blockDim * threadDim != matDim){
		std::cout << "Not enough/too many blocks and threads for given matrix dimensions" << std::endl;
		return 1;
	}

	// initalize more varaibles
	dim3 grid (blockDim, blockDim);
	dim3 block (threadDim, threadDim);

	//create arrays
	int *MatA = new int[(int)pow(matDim, 2)];
	int *MatB = new int[(int)pow(matDim, 2)]; 
	int *MatC = new int[(int)pow(matDim, 2)];

	for (int i=0; i < (int)pow(matDim, 2); i++) {
 		MatA[i] = i;
 		MatB[i] = i;
 	}

	//alloc memory
	int *a, *b, *c;
	hipMalloc( (void**)&a,(int)pow(matDim, 2) * sizeof(int) );
	hipMalloc( (void**)&b, (int)pow(matDim, 2) * sizeof(int) );
	hipMalloc( (void**)&c, (int)pow(matDim, 2) * sizeof(int) );

	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );

	//send to GPU
	hipMemcpy (a, MatA, (int)pow(matDim, 2) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy (b, MatB, (int)pow(matDim, 2) * sizeof(int), hipMemcpyHostToDevice);

	//add
	add <<<grid, block>>> (a, b, c);

	// get result from GPU
	hipMemcpy (MatC, c, (int)pow(matDim, 2) * sizeof(int), hipMemcpyDeviceToHost );

	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );

	//for testing output
	/*for (int i = 0; i < matDim; i++){
		for (int j = 0; j < matDim; j++){
			std::cout << MatC[(i*matDim)+j] << " ";
		}
		std::cout << std::endl;
	}*/

 	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	//dealloc memory
    	hipEventDestroy( start );
        hipEventDestroy( end );
	hipFree (a);
	hipFree (b);
	hipFree (c);
	delete MatA;
	MatA = NULL;
	delete MatB;
	MatB = NULL;
	delete MatC;
	MatC = NULL;
}
