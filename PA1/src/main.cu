#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "add.h"

bool isSquare(int num){	return (floor (sqrt(num)) == sqrt(num));}

int main (int argc, char* argv[]){
	
	//variables
	int matDim, blockDim, threadDim;

	// get inputs
	if (argc < 4){
		std::cout << "Not enough arguments" << std::endl; 
		return 1;
	}
	else{
	       matDim = atoi (argv [1]);
	       blockDim = atoi(argv [2]);
	       threadDim = atoi(argv [3]);
	}

	
	// bounds checking
	if ( matDim <=0){
		std::cout << "Matrix dimension not valid" << std::endl;
		return 1;
	}
	if ( blockDim <=0 ){
		std::cout << "Block dimension not valid" << std::endl;
		return 1;
	}
	if ( threadDim <=0 ){
		std::cout << "Matrix dimension not valid" << std::endl;
		return 1;
	}
	if ( blockDim * threadDim < matDim){
		std::cout << "Not enough blocks and threads for given matrix dimensions" << std::endl;
		return 1;
	}

	// initalize more varaibles
	dim3 grid (blockDim, blockDim);
	dim3 block (threadDim, threadDim);
	int addsPerThread = (int)pow(matDim, 2)/((int)pow(blockDim, 2)* (int)pow(threadDim, 2));
	//std::cout << addsPerThread << std::endl;

	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );


	//create arrays
	int *MatA = new int[(int)pow(blockDim, 2)* (int)pow(threadDim, 2)];
	int *MatB = new int[(int)pow(blockDim, 2)* (int)pow(threadDim, 2)]; 
	int *MatC = new int[(int)pow(blockDim, 2)* (int)pow(threadDim, 2)];

	for (int i=0; i < (int)pow(blockDim, 2)* (int)pow(threadDim, 2); i++) {
 		MatA[i] = i;
 		MatB[i] = i;
 	}

	//alloc memory
	int *a, *b, *c;
	hipMalloc( (void**)&a, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int) );
	hipMalloc( (void**)&b, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int) );
	hipMalloc( (void**)&c, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int) );

	//send to GPU
	hipMemcpy (a, MatA, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy (b, MatB, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int), hipMemcpyHostToDevice);

	//add
	add <<<grid, block>>> (a, b, c, addsPerThread);

	// get result from GPU
	hipMemcpy (MatC, c, pow(blockDim, 2)* pow(threadDim, 2) * sizeof(int), hipMemcpyDeviceToHost );

	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );

	for (int i = 0; i < matDim; i++){
		for (int j = 0; j < matDim; j++){
			std::cout << MatC[(i*matDim)+j] << " ";
		}
		std::cout << std::endl;
	}

 	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	//dealloc memory
    	hipEventDestroy( start );
        hipEventDestroy( end );
	hipFree (a);
	hipFree (b);
	hipFree (c);
	delete MatA;
	MatA = NULL;
	delete MatB;
	MatB = NULL;
	delete MatC;
	MatC = NULL;
}
