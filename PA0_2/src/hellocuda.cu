
#include <iostream>

#include "add.h"

int main() {
  
  int *dev_a, *dev_b, *dev_c;

  hipError_t err = hipMalloc( (void**) &dev_a, N * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  hipMallocManaged( (void**) &dev_a, N * sizeof(int));
  hipMallocManaged( (void**) &dev_b, N * sizeof(int));
  hipMallocManaged( (void**) &dev_c, N * sizeof(int));

  for (int i = 0; i < N; ++i) {
    dev_a[i] = i;
    dev_b[i] = i;
  }

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );

  add<<<N, 1>>>(dev_a, dev_b, dev_c);

  hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );

  for (int i = 0; i < N; ++i) {
    if (dev_c[i] != dev_a[i] + dev_b[i]) {
      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

      // clean up events - we should check for error codes here.
      hipEventDestroy( start );
      hipEventDestroy( end );

      // clean up device pointers - just like free in C. We don't have
      // to check error codes for this one.
      hipFree(dev_a);
      hipFree(dev_b);
      hipFree(dev_c);
      exit(1);
    }
  }

  std::cout << "Yay! Your program's results are correct." << std::endl;
  std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
  
  // Cleanup in the event of success.
  hipEventDestroy( start );
  hipEventDestroy( end );

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

}
