#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <cmath>

#include "mat.h"

bool isSquare(int num){	return (floor (sqrt(num)) == sqrt(num));}

int main (int argc, char* argv[]){
	
	//variables
	int matDim, blockDim, threadDim;

	// get inputs
	if (argc < 4){
		std::cout << "Not enough arguments. <<matrix dimension>> << block dimension>> << thread dimension>>" << std::endl; 
		return 1;
	}
	else{
	       matDim = atoi (argv [1]);
	       blockDim = atoi(argv [2]);
	       threadDim = atoi(argv [3]);
	}

	hipDeviceProp_t prop;
 	hipGetDeviceProperties( &prop, 0 );

	// bounds checking
	if ( matDim <=0 || matDim >= 32000){
		std::cout << "Matrix dimension not valid. Must be between 0 and 32000." << std::endl;
		return 1;
	}
	if ( blockDim <=0 || blockDim >= 25000 ){
		std::cout << "Block dimension not valid. Must be between 0 and 25000." << std::endl;
		return 1;
	}
	if ( threadDim <=0 || threadDim > sqrt(prop.maxThreadsPerBlock) ){
		std::cout << "Thread dimension not valid. Must be between 0 and " << sqrt(prop.maxThreadsPerBlock)  << "." << std::endl;
		return 1;
	}
	if ( blockDim * threadDim != matDim){
		std::cout << "Not enough/too many blocks and threads for given matrix dimensions" << std::endl;
		return 1;
	}

	// initalize more varaibles
	dim3 grid (blockDim, blockDim);
	dim3 block (threadDim, threadDim);

	//create arrays
	float *MatA = new float [(int)pow(matDim, 2)];
	float *MatB = new float [(int)pow(matDim, 2)]; 
	float *MatC = new float [(int)pow(matDim, 2)];

	for (int i=0; i < (int)pow(matDim, 2); i++) {
 		MatA[i] = i;
 		MatB[i] = i;
 	}

	//alloc memory
	float *a, *b, *c;
	hipMalloc( (void**)&a,(float)pow(matDim, 2) * sizeof(float) );
	hipMalloc( (void**)&b, (float)pow(matDim, 2) * sizeof(float) );
	hipMalloc( (void**)&c, (float)pow(matDim, 2) * sizeof(float) );

	// begin timing
 	hipEvent_t start, end;
  	hipEventCreate(&start);
  	hipEventCreate(&end);

 	hipEventRecord( start, 0 );

	//send to GPU
	hipMemcpy (a, MatA, (float)pow(matDim, 2) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy (b, MatB, (float)pow(matDim, 2) * sizeof(float), hipMemcpyHostToDevice);

	//add
	add <<<grid, block>>> (a, b, c);

	// get result from GPU
	hipMemcpy (MatC, c, (float)pow(matDim, 2) * sizeof(float), hipMemcpyDeviceToHost );

	//end time
	hipEventRecord( end, 0 );
  	hipEventSynchronize( end );

	//for testing output
	/*for (int i = 0; i < matDim; i++){
		for (int j = 0; j < matDim; j++){
			std::cout << MatC[(i*matDim)+j] << " ";
		}
		std::cout << std::endl;
	}*/

 	float elapsedTime;
  	hipEventElapsedTime( &elapsedTime, start, end );

        std::cout << "Time: " << elapsedTime << " ms." << std::endl;


	//dealloc memory
    	hipEventDestroy( start );
        hipEventDestroy( end );
	hipFree (a);
	hipFree (b);
	hipFree (c);
	delete MatA;
	MatA = NULL;
	delete MatB;
	MatB = NULL;
	delete MatC;
	MatC = NULL;
}
